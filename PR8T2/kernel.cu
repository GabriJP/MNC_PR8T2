/*
Resolver un sistema lineal con LAPACK en CPU y cuSOLVER en GPU
*/

#include <cstdio>
#include <random>
#include <mkl.h>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "eTimer.h"

#define N 2*1024

int main(int argc, char *argv[]) {
	std::random_device gen;
	std::normal_distribution<double> dist(0.0, 1.0);

	double *A1, *A2, *B1, *B2;

	A1 = static_cast<double*>(mkl_malloc(N * N * sizeof(double), 64));
	A2 = static_cast<double*>(mkl_malloc(N * N * sizeof(double), 64));
	B1 = static_cast<double*>(mkl_malloc(N * sizeof(double), 64));
	B2 = static_cast<double*>(mkl_malloc(N * sizeof(double), 64));

	for (auto i = 0; i < N; i++)
	{
		for (auto j = 0; j < N; j++)
		{
			A1[i*N + j] = dist(gen);
			A2[i*N + j] = A1[i*N + j];
		}
		B1[i] = dist(gen);
		B2[i] = B1[i];
		A1[i*N + i] += 10.0;
		A2[i*N + i] = A1[i*N + i];
	}

	int info;
	auto ipiv = static_cast<int*>(mkl_malloc(N * sizeof(double), 64));
	auto Tcpu = eTimer();
	auto Tgpu = eTimer();

	Tcpu.start();
	info = LAPACKE_dgesv(LAPACK_ROW_MAJOR, N, 1, A1, N, ipiv, B1, 1);
	Tcpu.stop();
	Tcpu.report("CPU");

	for (auto i = 0; i < 5; i++)
	{
		printf("%g ", B1[i]);
	}
	printf("\n\n");

	//Codigo de la GPU
	hipError_t cudaStatus;
	hipsolverStatus_t cusolverStatus;
	hipsolverHandle_t handle;

	double *dev_A, *dev_B, *Work;
	int *dev_pivot, *dev_info, Lwork;

	auto device = 0;
	cudaStatus = hipGetDevice(&device);
	cusolverStatus = hipsolverDnCreate(&handle);

	//Reservar espacio en GPU
	cudaStatus = hipMalloc(&dev_A, N*N * sizeof(double));
	cudaStatus = hipMalloc(&dev_B, N * sizeof(double));
	cudaStatus = hipMalloc(&dev_pivot, N * sizeof(int));
	cudaStatus = hipMalloc(&dev_info, sizeof(int));
	cudaStatus = hipMemcpy(dev_A, A2, N*N * sizeof(double), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_B, B2, N * sizeof(double), hipMemcpyHostToDevice);

	Tgpu.start();
	cusolverStatus = hipsolverDnDgetrf_bufferSize(handle, N, N, dev_A, N, &Lwork);
	cudaStatus = hipMalloc(&Work, Lwork * sizeof(double));
	cusolverStatus = hipsolverDnDgetrf(handle, N, N, dev_A, N, Work, dev_pivot, dev_info);
	cusolverStatus = hipsolverDnDgetrs(handle, HIPBLAS_OP_T, N, 1, dev_A, N, dev_pivot, dev_B, N, dev_info);
	cudaStatus = hipDeviceSynchronize();
	Tgpu.stop();
	Tgpu.report("GPU");

	cudaStatus = hipMemcpy(B2, dev_B, N * sizeof(double), hipMemcpyDeviceToHost);

	//Pero los datos recuperados est�n traspuestos
	for (auto i = 0; i < 5; i++)
	{
		printf("%g ", B2[i]);
	}
	printf("\n\n");

	//Liberaci�n de recursos
	cudaStatus = hipFree(dev_A);
	cudaStatus = hipFree(dev_B);
	cudaStatus = hipFree(dev_pivot);
	cudaStatus = hipFree(dev_info);
	cudaStatus = hipFree(Work);

	cusolverStatus = hipsolverDnDestroy(handle);

	cudaStatus = hipDeviceReset();
	//fin de la GPU

	mkl_free(A1);
	mkl_free(A2);
	mkl_free(B1);
	mkl_free(B2);
	mkl_free(ipiv);

	getchar();

	return 0;
}